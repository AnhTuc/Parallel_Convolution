// Nnah cho khoang kieu a.x+b.y+c.z( la 1 hang) --> (a+width).x+(b+width).x

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

__global__ void blurImgKernel(uchar3 * inPixels, int width, int height, 
		float * filter, int filterWidth, 
		uchar3 * outPixels)
{
	// TODO
	int r =blockIdx.y * blockDim.y + threadIdx.y;
	int c =blockIdx.x * blockDim.x + threadIdx.x;

	if(r>=height || c>=width) return;

	int idx=r*width+c;
	

	outPixels[idx].x=0.0;
	outPixels[idx].y=0.0;
	outPixels[idx].z=0.0;

	for(int i=0;i<filterWidth;i++){
		int cur_r=r-filterWidth/2 +i;

		if(cur_r<0) cur_r=0;
		if(cur_r>height-1) cur_r=height-1;

		for(int j=0;j<filterWidth;j++){
			int cur_c=c-filterWidth/2 +j;

			if(cur_c<0) cur_c=0;
			if(cur_c>width-1) cur_c=width-1;

			int mul=i*filterWidth+j;
			

			int cur=cur_r*width+cur_c;
			outPixels[idx].x+=filter[mul]*inPixels[cur].x;
			outPixels[idx].y+=filter[mul]*inPixels[cur].y;
			outPixels[idx].z+=filter[mul]*inPixels[cur].z;

			}
		}
	
}


void blurImg(uchar3 * inPixels, int width, int height, float * filter, int filterWidth, 
		uchar3 * outPixels,
		bool useDevice=false, dim3 blockSize=dim3(1, 1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// TODO
	
		for(int r=0;r<height;r++){
			for(int c=0;c<width;c++){
				int center=r*width+c;

				outPixels[center].x=0.0;
				outPixels[center].y=0.0;
				outPixels[center].z=0.0;

				for(int i=0;i<filterWidth;i++){
					int cur_r=r-filterWidth/2 +i;

					if(cur_r<0) cur_r=0;
					else if(cur_r>height-1) cur_r=height-1;

					for(int j=0; j<filterWidth;j++){
						int cur_c=c-filterWidth/2 +j;

						if(cur_c<0) cur_c=0;
						else if(cur_c>width-1) cur_c=width-1;

						int mul=i*filterWidth+j;

						int cur=cur_r*width+cur_c;
						outPixels[center].x+=inPixels[cur].x*filter[mul];
						outPixels[center].y+=inPixels[cur].y*filter[mul];
						outPixels[center].z+=inPixels[cur].z*filter[mul];
					}
				}
					
			}
		}

	}
	else // Use device
	{
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, 0);
		printf("GPU name: %s\n", devProp.name);
		printf("GPU compute capability: %d.%d\n", devProp.major, devProp.minor);

		// TODO
		// Allocate device memories
        uchar3 * d_in, * d_out;
		float * fil;
        size_t nBytes = width * height * sizeof(uchar3);
		size_t n_filter= filterWidth*filterWidth*sizeof(float);
        CHECK(hipMalloc(&d_in, nBytes));
        CHECK(hipMalloc(&fil, n_filter));
		CHECK(hipMalloc(&d_out, nBytes));

		// Copy data to device memories
        CHECK(hipMemcpy(d_in, inPixels, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(fil, filter, n_filter, hipMemcpyHostToDevice));


		// Set grid size and call kernel
        dim3 gridSize((width - 1) / blockSize.x + 1, 
                      (height - 1) / blockSize.y + 1);
        blurImgKernel<<<gridSize, blockSize >>>(d_in, width,height, fil,filterWidth,d_out);

		// Copy result from device memory
        CHECK(hipMemcpy(outPixels, d_out, nBytes, hipMemcpyDeviceToHost));

		// Free device memories
        CHECK(hipFree(d_in));
		CHECK(hipFree(fil));
        CHECK(hipFree(d_out));

	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n", 
    		useDevice == true? "use device" : "use host", time);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main(int argc, char ** argv)
{
	if (argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("Image size (width x height): %i x %i\n\n", width, height);

	// Read correct output image file
	int correctWidth, correctHeight;
	uchar3 * correctOutPixels;
	readPnm(argv[3], correctWidth, correctHeight, correctOutPixels);
	if (correctWidth != width || correctHeight != height)
	{
		printf("The shape of the correct output image is invalid\n");
		return EXIT_FAILURE;
	}

	// Set up a simple filter with blurring effect 
	int filterWidth = 9;
	float * filter = (float *)malloc(filterWidth * filterWidth * sizeof(float));
	for (int filterR = 0; filterR < filterWidth; filterR++)
	{
		for (int filterC = 0; filterC < filterWidth; filterC++)
		{
			filter[filterR * filterWidth + filterC] = 1. / (filterWidth * filterWidth);
		}
	}

	// Blur input image using host
	uchar3 * hostOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3)); 
	blurImg(inPixels, width, height, filter, filterWidth, hostOutPixels);
	
	// Compute mean absolute error between host result and correct result
	float hostErr = computeError(hostOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", hostErr);

	// Blur input image using device
	uchar3 * deviceOutPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	dim3 blockSize(32, 32); // Default
	if (argc == 6)
	{
		blockSize.x = atoi(argv[4]);
		blockSize.y = atoi(argv[5]);
	}  
	blurImg(inPixels, width, height, filter, filterWidth, deviceOutPixels, true, blockSize);

	// Compute mean absolute error between device result and correct result
	float deviceErr = computeError(deviceOutPixels, correctOutPixels, width * height);
	printf("Error: %f\n\n", deviceErr);

	// Write results to files
	char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(hostOutPixels, width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(deviceOutPixels, width, height, concatStr(outFileNameBase, "_device.pnm"));

	// Free memories
	free(inPixels);
	free(correctOutPixels);
	free(hostOutPixels);
	free(deviceOutPixels);
	free(filter);
}
